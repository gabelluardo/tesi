#include "hip/hip_runtime.h"
__global__ void kernelFn(int *vec_a, int *vec_b, int *res, uint n) {
  uint i = threadIdx.x + blockDim.x * blockIdx.x;

  if (i < n) {
    res[i] = vec_a[i] + vec_b[i];
  }
}

int main(int argc, char **argv) {
  uint size = len * sizeof(int);
  int *h_vec_a = reinterpret_cast<int *>(malloc(size));
  int *h_vec_b = reinterpret_cast<int *>(malloc(size));
  int *h_res_vec = reinterpret_cast<int *>(malloc(size));

  int *vec_a, *vec_b, *res_vec;

  hipMalloc(&vec_a, size);
  hipMemcpy(vec_a, h_vec_a, size, hipMemcpyHostToDevice);
  hipMalloc(&vec_b, size);
  hipMemcpy(vec_b, h_vec_b, size, hipMemcpyHostToDevice);

  hipMalloc(&res_vec, size);

  kernelFn<<<ceil(len / 64.0), 64>>>(vec_a, vec_b, res_vec, len);
  hipDeviceSynchronize();

  hipMemcpy(h_res_vec, res_vec, size, hipMemcpyDeviceToHost);

  // ...

  hipFree(res_vec);
  hipFree(vec_b);
  hipFree(vec_a);
  free(h_res_vec);
  free(h_vec_b);
  free(h_vec_a);

  return 0;
}